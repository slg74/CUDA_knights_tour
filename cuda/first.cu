
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
    printf("Hello from GPU!\n"); 
}

int main(void) {
    printf("Hello from GPU!\n"); 

    helloFromGPU <<<1, 10>>>();
    hipDeviceReset();
    return 0;
}
